/*
Copyright (c) 2023 Advanced Micro Devices, Inc. All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/

#include "stdio.h"

#ifndef ENABLE_HIP
#include <hip/hip_runtime.h>
#else
// Translate CUDA functions to the analogous HIP function
#include "hipifly.h"
#endif

#define TPB 256 //Number of threads per block for kernel calls

void get_device_properties( const int device_id ){
 	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, device_id );
  printf("Device: %d  name:  %s \n", device_id, prop.name );

}


int set_device( int device_id ){
  
  int n_devices;
  hipGetDeviceCount(&n_devices);
  printf("Number of available devices %d\n", n_devices);  
  printf("Device id: %d \n", device_id);
  if ( device_id >= n_devices ){
    printf( "ERROR: Device %d is not available. Only %d devices detected.\n", device_id, n_devices );
    return -1;
  }
  
  hipSetDevice(device_id); 
  get_device_properties( device_id );

  return 0;

}

void allocate_device_arrays( int N, double *&d_a, double *&d_b, double *&d_c  ){
  hipMalloc( (void **)&d_a, N*sizeof(double) );
  hipMalloc( (void **)&d_b, N*sizeof(double) );
  hipMalloc( (void **)&d_c, N*sizeof(double) );  
}


void copy_host_to_device( int N, double *h_a, double *h_b, 
                          double *&d_a, double *&d_b   ){
  hipMemcpy( d_a, h_a, N*sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy( d_b, h_b, N*sizeof(double), hipMemcpyHostToDevice );
}


__global__ void vector_add_kernel( int N, double *d_a, double *d_b, double *d_c ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if ( tid < N ){
    d_c[tid] = d_a[tid] + d_b[tid];
  }
}

float gpu_vector_add( int N, double *d_a, double *d_b, double *d_c ){

  int n_grid = ( N - 1 )/TPB + 1;
  dim3 grid( n_grid, 1, 1 );
  dim3 block( TPB, 1, 1 );

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  vector_add_kernel<<<grid, block, 0, 0>>>( N, d_a, d_b, d_c );
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  
  float elapsed_time_milliseconds = 0;
  hipEventElapsedTime(&elapsed_time_milliseconds, start, stop);
  return elapsed_time_milliseconds;

}


void copy_device_to_host( int N, double *d_a, double *h_a ){
  hipMemcpy( h_a, d_a, N*sizeof(double), hipMemcpyDeviceToHost);
}
